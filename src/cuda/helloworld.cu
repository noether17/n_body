
#include <hip/hip_runtime.h>
#include "stdio.h"

#define WARPSIZE 32

__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x;
    if (tid < WARPSIZE)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void)
{
    int a[WARPSIZE];
    int b[WARPSIZE];
    int c[WARPSIZE];
    for (int i = 0; i < WARPSIZE; i++)
    {
        a[i] = i;
        b[i] = i + WARPSIZE;
    }

    int* dev_a;
    int* dev_b;
    int* dev_c;
    hipMalloc((void**)&dev_a, WARPSIZE * sizeof(int));
    hipMalloc((void**)&dev_b, WARPSIZE * sizeof(int));
    hipMalloc((void**)&dev_c, WARPSIZE * sizeof(int));
    hipMemcpy(dev_a, a, WARPSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, WARPSIZE * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, WARPSIZE>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, WARPSIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for (int i = 0; i < WARPSIZE; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}