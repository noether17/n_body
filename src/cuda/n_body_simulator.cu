
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"
#include "sys/time.h"

const double G = 6.67408e-11; // gravitational constant in m^3 kg^-1 s^-2
const double L = 1.0; // box width in m
const double epsilon = 1e-3*L; // softening parameter in m
const double softening2 = epsilon*epsilon;

struct Vector3d
{
    double x;
    double y;
    double z;
};

__global__ void update_acceleration(Vector3d* acc, Vector3d* pos, int n)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n)
    {
        for (int j = 0; j < n; ++j)
        {
            if (i != j)
            {
                double dx = pos[j].x - pos[i].x;
                double dy = pos[j].y - pos[i].y;
                double dz = pos[j].z - pos[i].z;
                double r2 = dx*dx + dy*dy + dz*dz;
                double denominator = (r2 + softening2)*sqrt(r2);
                acc[i].x += G*dx / denominator;
                acc[i].y += G*dy / denominator;
                acc[i].z += G*dz / denominator;
            }
        }
    }
}

__global__ void update_state(Vector3d* pos, Vector3d* vel, Vector3d* acc, double dt, int n)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n)
    {
        pos[i].x += vel[i].x*dt;
        pos[i].y += vel[i].y*dt;
        pos[i].z += vel[i].z*dt;
        vel[i].x += acc[i].x*dt;
        vel[i].y += acc[i].y*dt;
        vel[i].z += acc[i].z*dt;
    }
}

struct OutputEntry
{
    double t;
    size_t index;
    Vector3d pos;
    Vector3d vel;
};

__global__ void output_states(OutputEntry* out_states, Vector3d* pos, Vector3d* vel, int n,
    int step_index, double t)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int step_offset = step_index*n;
    if (i < n)
    {
        out_states[step_offset + i].t = t;
        out_states[step_offset + i].index = (size_t)i;
        out_states[step_offset + i].pos.x = pos[i].x;
        out_states[step_offset + i].pos.y = pos[i].y;
        out_states[step_offset + i].pos.z = pos[i].z;
        out_states[step_offset + i].vel.x = vel[i].x;
        out_states[step_offset + i].vel.y = vel[i].y;
        out_states[step_offset + i].vel.z = vel[i].z;
    }
}

//__global__ void print_output_states(OutputEntry* out_states, int n, int n_steps)
//{
//    int i = blockIdx.x*blockDim.x + threadIdx.x;
//    if (i < n)
//    {
//        for (int j = 0; j < n_steps; ++j)
//        {
//            OutputEntry* entry = &out_states[j*n + i];
//            printf("i = %d, t = %.16e, index = %lu, pos.x = %.16e, pos.y = %.16e, pos.z = %.16e, vel.x = %.16e, vel.y = %.16e, vel.z = %.16e\n",
//                i, entry->t, entry->index,
//                entry->pos.x, entry->pos.y, entry->pos.z,
//                entry->vel.x, entry->vel.y, entry->vel.z);
//        }
//    }
//}

double cuda_euler_loop(Vector3d* pos, Vector3d* vel, int n, double dt, double max_time,
    OutputEntry** out_states, size_t* out_nstates)
{
    Vector3d* d_pos;
    Vector3d* d_vel;
    Vector3d* d_acc;
    hipMalloc(&d_pos, n*sizeof(Vector3d));
    hipMalloc(&d_vel, n*sizeof(Vector3d));
    hipMalloc(&d_acc, n*sizeof(Vector3d));
    hipMemcpy(d_pos, pos, n*sizeof(Vector3d), hipMemcpyHostToDevice);
    hipMemcpy(d_vel, vel, n*sizeof(Vector3d), hipMemcpyHostToDevice);
    int n_reserve_steps = (int)(max_time / dt) * 2;
    OutputEntry* d_out_states;
    hipMalloc(&d_out_states, n_reserve_steps*n*sizeof(OutputEntry));
    int block_size = 256;
    int num_blocks = (n + block_size - 1) / block_size;
    int n_steps = 0;
    struct timeval start;
    gettimeofday(&start, NULL);
    for (double t = 0.0; t < max_time; t += dt)
    {
        update_acceleration<<<num_blocks, block_size>>>(d_acc, d_pos, n);
        update_state<<<num_blocks, block_size>>>(d_pos, d_vel, d_acc, dt, n);

        output_states<<<num_blocks, block_size>>>(d_out_states, d_pos, d_vel, n, n_steps, t);
        ++n_steps;
    }
    struct timeval end;
    gettimeofday(&end, NULL);
    double elapsed_time = (end.tv_sec - start.tv_sec) + 1e-6*(end.tv_usec - start.tv_usec);
    OutputEntry* out_states_host = (OutputEntry*)malloc(n_steps*n*sizeof(OutputEntry));
    hipMemcpy(out_states_host, d_out_states, n_steps*n*sizeof(OutputEntry), hipMemcpyDeviceToHost);
    hipMemcpy(pos, d_pos, n*sizeof(Vector3d), hipMemcpyDeviceToHost);
    hipMemcpy(vel, d_vel, n*sizeof(Vector3d), hipMemcpyDeviceToHost);
    hipFree(d_out_states);
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(d_acc);

    *out_states = out_states_host;
    *out_nstates = n_steps*n;

    return elapsed_time;
}

void output_results(const char* filename, OutputEntry* out_states, size_t nstates)
{
    FILE* fp = fopen(filename, "w");
    for (size_t i = 0; i < nstates; ++i)
    {
        OutputEntry* entry = &out_states[i];
        fprintf(fp, "%lf,%lu,%lf,%lf,%lf,%lf,%lf,%lf\n",
            entry->t, entry->index,
            entry->pos.x, entry->pos.y, entry->pos.z,
            entry->vel.x, entry->vel.y, entry->vel.z);
    }
    fclose(fp);
}

int N = 1 << 3;
int output_to_file = 0;

int main(int argc, char** argv)
{
    if (argc > 1) { N = atoi(argv[1]); }
    if (argc > 2) { output_to_file = atoi(argv[2]); }

    // initialize state
    Vector3d* pos = (Vector3d*)malloc(N*sizeof(Vector3d));
    Vector3d* vel = (Vector3d*)malloc(N*sizeof(Vector3d));
    for (int i = 0; i < N; ++i)
    {
        pos[i].x = L*rand() / RAND_MAX;
        pos[i].y = L*rand() / RAND_MAX;
        pos[i].z = L*rand() / RAND_MAX;
        vel[i].x = 0.0;
        vel[i].y = 0.0;
        vel[i].z = 0.0;
    }

    // run simulation
    double max_time = sqrt(L*L*L / (G * N));
    double dt = 1e-3*max_time;
    OutputEntry* out_states;
    size_t nstates;
    struct timeval start;
    gettimeofday(&start, NULL);
    double time_on_gpu = cuda_euler_loop(pos, vel, N, dt, max_time, &out_states, &nstates);
    struct timeval end;
    gettimeofday(&end, NULL);
    double total_seconds = (end.tv_sec - start.tv_sec) + 1e-6*(end.tv_usec - start.tv_usec);
    printf("N = %d, time on GPU = %lfs, total time = %lfs\n", N, time_on_gpu, total_seconds);

    // output results
    if (output_to_file)
    {
        char filename[256];
        sprintf(filename, "cudaoutput_%d_.csv", N);
        output_results(filename, out_states, nstates);
    }
    free(out_states);

    // free state
    free(pos);
    free(vel);

    return 0;
}