
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"

const double G = 6.67408e-11; // gravitational constant in m^3 kg^-1 s^-2
const double L = 1.0; // box width in m
const double epsilon = 1e-3*L; // softening parameter in m
const double softening2 = epsilon*epsilon;

struct Vector3d
{
    double x;
    double y;
    double z;
};

__global__ void update_acceleration(Vector3d* acc, Vector3d* pos, int n)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n)
    {
        double ax = 0.0;
        double ay = 0.0;
        double az = 0.0;
        for (int j = 0; j < n; ++j)
        {
            if (i != j)
            {
                double dx = pos[j].x - pos[i].x;
                double dy = pos[j].y - pos[i].y;
                double dz = pos[j].z - pos[i].z;
                double r2 = dx*dx + dy*dy + dz*dz;
                double denominator = (r2 + softening2)*sqrt(r2);
                ax += dx / denominator;
                ay += dy / denominator;
                az += dz / denominator;
            }
        }
        acc[i].x = G*ax;
        acc[i].y = G*ay;
        acc[i].z = G*az;
    }
}

__global__ void update_state(Vector3d* pos, Vector3d* vel, Vector3d* acc, double dt, int n)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n)
    {
        pos[i].x += vel[i].x*dt;
        pos[i].y += vel[i].y*dt;
        pos[i].z += vel[i].z*dt;
        vel[i].x += acc[i].x*dt;
        vel[i].y += acc[i].y*dt;
        vel[i].z += acc[i].z*dt;
    }
}

struct OutputEntry
{
    double t;
    size_t index;
    Vector3d pos;
    Vector3d vel;
};

__global__ void output_states(OutputEntry* out_states, Vector3d* pos, Vector3d* vel, int n,
    size_t step_index, double t)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int step_offset = step_index*n;
    if (i < n)
    {
        out_states[step_offset + i].t = t;
        out_states[step_offset + i].index = i;
        out_states[step_offset + i].pos = pos[i];
        out_states[step_offset + i].vel = vel[i];
    }
}

void cuda_euler_loop(Vector3d* pos, Vector3d* vel, int n, double dt, double max_time,
    OutputEntry** out_states, size_t* out_nstates)
{
    Vector3d* d_pos;
    Vector3d* d_vel;
    Vector3d* d_acc;
    hipMalloc(&d_pos, n*sizeof(Vector3d));
    hipMalloc(&d_vel, n*sizeof(Vector3d));
    hipMalloc(&d_acc, n*sizeof(Vector3d));
    hipMemcpy(d_pos, pos, n*sizeof(Vector3d), hipMemcpyHostToDevice);
    hipMemcpy(d_vel, vel, n*sizeof(Vector3d), hipMemcpyHostToDevice);
    int n_reserve_steps = (int)(max_time / dt + 1.0);
    OutputEntry* d_out_states;
    hipMalloc(&d_out_states, n_reserve_steps*n*sizeof(OutputEntry));
    int block_size = 256;
    int num_blocks = (n + block_size - 1) / block_size;
    int n_steps = 0;
    for (double t = 0.0; t < max_time; ++t)
    {
        update_acceleration<<<num_blocks, block_size>>>(d_acc, d_pos, n);
        update_state<<<num_blocks, block_size>>>(d_pos, d_vel, d_acc, dt, n);

        output_states<<<num_blocks, block_size>>>(d_out_states, d_pos, d_vel, n, n_steps, t);
        ++n_steps;
    }
    OutputEntry* out_states_host = (OutputEntry*)malloc(n_steps*n*sizeof(OutputEntry));
    hipMemcpy(out_states_host, d_out_states, n_steps*n*sizeof(OutputEntry), hipMemcpyDeviceToHost);
    hipMemcpy(pos, d_pos, n*sizeof(Vector3d), hipMemcpyDeviceToHost);
    hipMemcpy(vel, d_vel, n*sizeof(Vector3d), hipMemcpyDeviceToHost);
    hipFree(d_out_states);
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(d_acc);

    *out_states = out_states_host;
    *out_nstates = n_steps*n;
}

void output_results(const char* filename, OutputEntry* out_states, size_t nstates)
{
    FILE* fp = fopen(filename, "w");
    for (size_t i = 0; i < nstates; ++i)
    {
        OutputEntry* entry = &out_states[i];
        fprintf(fp, "%f,%zu,%f,%f,%f,%f,%f,%f\n",
            entry->t, entry->index,
            entry->pos.x, entry->pos.y, entry->pos.z,
            entry->vel.x, entry->vel.y, entry->vel.z);
    }
    fclose(fp);
}

int N = 1 << 3;

int main(int argc, char** argv)
{
    if (argc > 1) { N = atoi(argv[1]); }

    // initialize state
    Vector3d* pos = (Vector3d*)malloc(N*sizeof(Vector3d));
    Vector3d* vel = (Vector3d*)malloc(N*sizeof(Vector3d));
    for (int i = 0; i < N; ++i)
    {
        pos[i].x = L*rand() / RAND_MAX;
        pos[i].y = L*rand() / RAND_MAX;
        pos[i].z = L*rand() / RAND_MAX;
        vel[i].x = 0.0;
        vel[i].y = 0.0;
        vel[i].z = 0.0;
    }

    // run simulation
    double max_time = sqrt(L*L*L / (G * N));
    double dt = 1e-3*max_time;
    OutputEntry* out_states;
    size_t nstates;
    cuda_euler_loop(pos, vel, N, dt, max_time, &out_states, &nstates);

    // output results
    output_results("nbody_cuda.csv", out_states, nstates);

    // free state
    free(pos);
    free(vel);

    return 0;
}